﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__device__ __managed__ u32 gtime = 0;


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
  
  //initial the bits-map
  for(int i=0; i < fs->STORAGE_SIZE; i++){
	  fs->volume[i] = uchar(255);
  }
  
  for(int i=0; i < fs->SUPERBLOCK_SIZE; i++){
	  fs->volume[i] = uchar(0);
  }

}

/*
 * my FCB structure
 * |0|1|2|3|4|5|6|7|8|9|10|11|12|13|14|15|16|17|18|19|   20|21   | 22|23 | 24 | 25|26  | 27|28  |
 * |                    file name                    |  location |  size |mode|create_t|modify_t|
 */

__device__ int string_len(char*s) {
	//方程的返回值是包含了\0的数学长度
	int str_len = 0;
	while (true) {
		str_len += 1;
		if (s[str_len] == '\0') {
			break;
		}
	}
	return str_len+1;
}

__device__ int find_first_free_block_offset(FileSystem *fs){
	//函数返回的是 第几个block是free的，offset是从0开始计算的

	int free_bit_map_index;
	int bitmap_num;
	int free_block_offset;
	for(int k=0;k<fs->SUPERBLOCK_SIZE;k++){
		if(fs->volume[k] < 255){
			free_bit_map_index = k;
			bitmap_num = int(fs->volume[k]);
			break;
		}
	}
	//通过bitmap number，确定到底第几个block是空的。应该有八种情况
	if(bitmap_num == 0){
		//0000 0000
		free_block_offset = 8*free_bit_map_index;				
	}else if(bitmap_num == 128){
		//1000 0000
		free_block_offset = 8*free_bit_map_index + 1;
	}else if(bitmap_num == 192){
		//1100 0000
		free_block_offset = 8*free_bit_map_index + 2;
	}else if(bitmap_num == 224){
		//1110 0000
		free_block_offset = 8*free_bit_map_index + 3;
	}else if(bitmap_num == 240){
		//1111 0000
		free_block_offset = 8*free_bit_map_index + 4;
	}else if(bitmap_num == 248){
		//1111 1000
		free_block_offset = 8*free_bit_map_index + 5;
	}else if(bitmap_num == 252){
		//1111 1100
		free_block_offset = 8*free_bit_map_index + 6;
	}else if(bitmap_num == 254){
		//1111 1110
		free_block_offset = 8*free_bit_map_index + 7;
	}
	return free_block_offset;
}

__device__ int trans_num(int a) {
	int result;
	if (a == 0) {
		result = 0;
	}
	else if (a == 128) {
		result = 1;
	}
	else if (a == 192) {
		result = 2;
	}
	else if (a == 224) {
		result = 3;
	}
	else if (a == 240) {
		result = 4;
	}
	else if (a == 248) {
		result = 5;
	}
	else if (a == 252) {
		result = 6;
	}
	else if (a == 254) {
		result = 7;
	}
	else if (a == 255) {
		result = 8;
	}
	return result;
}

__device__ void change_bitmaps(FileSystem *fs, int block_num){
	int bit_exits = 0; 
	int free_bit_map_index;
	
	for (int i = 0; i < fs->SUPERBLOCK_SIZE; i++) {
		
		if (int(fs->volume[i]) == uchar(0)) {
			break;
		}
		
		bit_exits += trans_num(int(fs->volume[i]));
		
	}
	for (int i = 0; i < fs->SUPERBLOCK_SIZE; i++) {
		if (int(fs->volume[i]) < uchar(255)) {
			free_bit_map_index = i;
			//printf("free_bit_map_index is %d\n", free_bit_map_index);
			break;
		}
	}
	//printf("bits_exits is %d\n",bit_exits);
	int exits_byte_num = bit_exits/8;
	int exits_bit_num = bit_exits%8;
	
	int bit_total = bit_exits + block_num;
	int total_byte_num = bit_total/8;
	int total_bit_num = bit_total%8;
	int last_index;
	
	
	
	if(total_byte_num - exits_byte_num == 0){
		last_index = free_bit_map_index;
	}else{
		int distance = total_byte_num-exits_byte_num;
		for(int j=0; j<distance; j++){
			fs->volume[free_bit_map_index + j] = 255;
		}
		last_index = free_bit_map_index+distance;
		
	}
	if(total_bit_num == 1){
		fs->volume[last_index] = 128;
	}else if(total_bit_num == 2){
		fs->volume[last_index] = 192;
	}else if(total_bit_num == 3){
		fs->volume[last_index] = 224;
	}else if(total_bit_num == 4){
		fs->volume[last_index] = 240;
	}else if(total_bit_num == 5){
		fs->volume[last_index] = 248;
	}else if(total_bit_num == 6){
		fs->volume[last_index] = 252;
	}else if(total_bit_num == 7){
		fs->volume[last_index] = 254;
	}
}

__device__ int same_block_size(int orig_size,int size){
	int orig_block = orig_size/32;
	if(orig_size%32>0){
		orig_block += 1;
	}
	
	int curr_block = size/32;
	if(size%32 >0){
		curr_block += 1;
	}
	if(orig_block == curr_block){
		return 1;
	}else if(orig_block < curr_block){
		return 2;
	}else if(orig_block > curr_block){
		return 3;
	}
}

__device__ int sizeToBlock(int size){
	int curr_block = size/32;

	if(size%32 >0){
		curr_block += 1;
	}
	return curr_block;
}


__device__ int find_free_bitmap_index(FileSystem *fs){
	for(int j=0;j<fs->SUPERBLOCK_SIZE;j++){
		if(int(fs->volume[j])<255){
			int free_byte = j;
			return free_byte;
		}
	}
}

__device__ void compact(FileSystem *fs, int block_start_offset, int block_start_index, int orig_size){
	//printf("first bit map is %d\n", int(fs->volume[0]));
	int free_block_offset = find_first_free_block_offset(fs);  //找到第一空的block的offset
	//printf("free block offset is %d\n", free_block_offset);

	int orig_block = sizeToBlock(orig_size); //original size占用多少blocks
	//printf("orig_block is %d\n", orig_block);

	int compact_content_block_num = free_block_offset - block_start_offset - orig_block;
	int orig_byte_size = orig_block*32;
	for(int i=0 ; i< compact_content_block_num*32; i++){
		fs->volume[block_start_index+i] = fs->volume[block_start_index + orig_byte_size+i];
	}
	//printf("here1");
	for(int k=0 ; k<orig_byte_size; k++){
		fs->volume[block_start_index+ compact_content_block_num*32+k] = 255;
	}
	//printf("here2");
	
	//compact bitmap
	int count = orig_block;
	int last_free_byte;
	int last_num;
	int last_block;
	int dis;
	while(count>=0){
		//printf("count!\n");
		last_free_byte = find_free_bitmap_index(fs);
		last_num = int(fs->volume[last_free_byte]);
		//printf("last num is %d\n", last_num);
		if(last_num == 0){
			last_block = 0;
		}else if(last_num = 128){
			last_block = 1;
		}else if(last_num = 192){
			last_block = 2;
		}else if(last_num = 224){
			last_block = 3;
		}else if(last_num = 240){
			last_block = 4;
		}else if(last_num = 248){
			last_block = 5;
		}else if(last_num = 252){
			last_block = 6;
		}else if(last_num = 254){
			last_block = 7;
		}
		//printf("last block is %d\n", last_block);
		if (last_block >= count){       //如果bitmap最后的byte里面存放的blocks足够满足origin的需求
			dis = last_block - count;
			if(dis == 1){
				fs->volume[last_free_byte] = uchar(128);
			}else if(dis == 2){
				fs->volume[last_free_byte] = uchar(192);
			}else if(dis == 3){
				fs->volume[last_free_byte] = uchar(224);
			}else if(dis == 4){
				fs->volume[last_free_byte] = uchar(240);
			}else if(dis == 5){
				fs->volume[last_free_byte] = uchar(248);
			}else if(dis == 6){
				fs->volume[last_free_byte] = uchar(252);
			}else if(dis == 7){
				fs->volume[last_free_byte] = uchar(254);
			}
			break;
		}else{                 //如果bitmap最后的byte不够满足这个需求
			//last_block < count
			fs->volume[last_free_byte] = uchar(0);
			fs->volume[last_free_byte-1] = uchar(254);
			count = count - last_block -1;
		}
	}
}

__device__ int find_last_fcb(FileSystem *fs){
	//return的是最后一个有元素的fcb的index
	int count = -1;
	int last_index;
	for(int i=0; i<1024; i++){
		if(fs->volume[fs->SUPERBLOCK_SIZE+32*i] == uchar(255)){
			count = i;
			break;
		}
	}
	if(count >=0){
		last_index = fs->SUPERBLOCK_SIZE + 32*(count-1);
	}else{
		last_index = fs->FILE_BASE_ADDRESS - 32;     //fcb的最后一个block就是最后一个有元素的index
	}
	return last_index;
}

__device__ int bubble_sort(FileSystem *fs, int op){
	int last_fcb_index = find_last_fcb(fs);          //找到最后一个有元素的fcb的实际index
	int fcb_num = (last_fcb_index - fs->SUPERBLOCK_SIZE)/32 + 1;  //一共有多少个fcb，数学意义上的
	int target1;
	int target2;
	uchar temp[32];
	
	if(op == LS_D){
		target1 = 27;
		target2 = 28;
		for(int i=0; i< fcb_num-1; i++){
			//printf("bubble sort times %d\n",i );
			for(int j=0; j<fcb_num-1-i; j++){
				if(int((fs->volume[fs->SUPERBLOCK_SIZE + 32*j + target1]<<8) + fs->volume[fs->SUPERBLOCK_SIZE + 32*j + target2] ) < int((fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + target1]<<8) + fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + target2] )){
					for (int x=0; x<32; x++){
						temp[x] = fs->volume[fs->SUPERBLOCK_SIZE + 32*j + x];
					}
					for(int y=0; y<32; y++){
						fs->volume[fs->SUPERBLOCK_SIZE + 32*j + y] = fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + y];
					}
					for(int z=0; z<32; z++){
						fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + z] = temp[z];
					}
				}
			}
		}
		
	}else if(op == LS_S){
		target1 = 22;
		target2 = 23;
		for(int i=0; i< fcb_num-1; i++){
			//printf("bubble sort times %d\n", i);
			for(int j=0; j<fcb_num-1-i; j++){
				if(int((fs->volume[fs->SUPERBLOCK_SIZE + 32*j + target1]<<8) + fs->volume[fs->SUPERBLOCK_SIZE + 32*j + target2] ) < int((fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + target1]<<8) + fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + target2] )){
					for (int x=0; x<32; x++){
						temp[x] = fs->volume[fs->SUPERBLOCK_SIZE + 32*j + x];
					}
					for(int y=0; y<32; y++){
						fs->volume[fs->SUPERBLOCK_SIZE + 32*j + y] = fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + y];
					}
					for(int z=0; z<32; z++){
						fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + z] = temp[z];
					}
				}else if(int((fs->volume[fs->SUPERBLOCK_SIZE + 32*j + target1]<<8) + fs->volume[fs->SUPERBLOCK_SIZE + 32*j + target2] ) == int((fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + target1]<<8) + fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + target2] )){
					if(int((fs->volume[fs->SUPERBLOCK_SIZE + 32*j + 25]<<8) + fs->volume[fs->SUPERBLOCK_SIZE + 32*j + 26]) > int((fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + 25]<<8) + fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + 26] )){
						for (int x=0; x<32; x++){
							temp[x] = fs->volume[fs->SUPERBLOCK_SIZE + 32*j + x];
						}
						for(int y=0; y<32; y++){
							fs->volume[fs->SUPERBLOCK_SIZE + 32*j + y] = fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + y];
						}
						for(int z=0; z<32; z++){
							fs->volume[fs->SUPERBLOCK_SIZE + 32*(j+1) + z] = temp[z];
						}
					}
				}
			}
		}
	}
	return fcb_num;
}

__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{	
	//open 函数return的是FCB起始点的实际index
	//printf("This is open function, the length of the file name is %d\n", string_len(s));
	gtime++;
	if (string_len(s)>20){
		printf("ERROR: The length of the file name exceeds the maximum!\n");
		return -1;
	}
	int fcb_index;
	int fcb_find_offset = -3;   //denote the position of the target file in fcb
	int fcb_free_offset = -1;
	int free_bit_map_index=-1;
	int compare_flag = 1;   //1 indicates they are equal, 0 indicates they are different
	int bitmap_num;
	int free_block_offset;
	if(op == G_WRITE){
		//write mode
		for(int i=0; i<1024;i++){             //遍历所有的FCB，找到是否有匹配的名字
			compare_flag = 1;
			fcb_index = fs->SUPERBLOCK_SIZE+i*fs->FCB_SIZE;
			if(fs->volume[fcb_index] == 255){
				//如果没满 但是一定找不到了
				fcb_find_offset = -2;    //did not find file name s in fcb
				fcb_free_offset = i;
				break;
			}
			else{
				//这个不是空的
				for(int j=0; j<string_len(s);j++){
					if(fs->volume[fcb_index + j] != s[j]){
						compare_flag = 0;
						break;
					}
				}
				if(compare_flag == 1){
					//we find the file name
					fcb_find_offset = i;
					break;
				}
			}
		}
		if(fcb_find_offset >=0){
			//find one in fcb
			fcb_index = fs->SUPERBLOCK_SIZE + fcb_find_offset*fs->FCB_SIZE;
			fs->volume[fcb_index + 24] = uchar('w');  //change the mode to write
			return fcb_index;                   //return the actual index of the file information in fcb
		}
		else if(fcb_find_offset == -3){
			printf("ERROR: File system is full, file name does not present!\n");
			return -1;
		}
		else if(fcb_find_offset == -2){
			//fbc没满但是没找到name

			free_block_offset = find_first_free_block_offset(fs);			
			
			fcb_index = fs->SUPERBLOCK_SIZE + fs->FCB_SIZE*fcb_free_offset;
			fs->volume[fcb_index + 24] = uchar('w');
			for (int l=0;l< string_len(s);l++){
				fs->volume[fcb_index + l] = s[l];
			}
			fs->volume[fcb_index + 20] = uchar(free_block_offset >>8);
			fs->volume[fcb_index + 21] = uchar(free_block_offset & 0x000000FF);
			fs->volume[fcb_index + 25] = uchar(gtime >>8);
			fs->volume[fcb_index + 26] = uchar(gtime & 0x000000FF);
			return fcb_index;
		}
	}
	else if(op == G_READ){
		//read mode
		for(int i=0; i<1024;i++){
			compare_flag = 1;
			fcb_index = fs->SUPERBLOCK_SIZE+i*fs->FCB_SIZE;
			if(fs->volume[fcb_index] == 255){
				fcb_find_offset = -2;    //did not find file name s in fcb
				break;
			}
			else{
				for(int j=0; j< string_len(s);j++){
					if(fs->volume[fcb_index + j] != s[j]){
						compare_flag = 0;
						break;
					}
				}
				if(compare_flag == 1){
					fcb_find_offset = i;
					break;
				}
			}
		}
		if(fcb_find_offset == -3 || fcb_find_offset==-2){
			printf("ERROR: The file is not in the file system, cannot read it \n");
			return -1;
		}else{
			//printf("fcb_find_offset is %d\n", fcb_find_offset);
			fcb_index = fs->SUPERBLOCK_SIZE+fs->FCB_SIZE*fcb_find_offset;
			//printf("fcb_index is %d\n", fcb_index);
			fs->volume[fcb_index+24] = uchar('r');
			return fcb_index;
		}
	}
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	if(fs->volume[fp+24]== uchar('w')){
		printf("ERROR: Cannnot implement read on write file!\n");
		return;
	}
	//printf("fcb_index is %d\n", fp);
	int block_offset = int((fs->volume[fp+20]<<8) + fs->volume[fp+21]);
	//printf("block_offset is %d\n", block_offset);
	int block_addr = fs->FILE_BASE_ADDRESS + fs->STORAGE_BLOCK_SIZE *block_offset;
	//printf("block_addr is %d\n", block_addr);
	
	for(int i=0; i<size; i++){
		output[i] = fs->volume[block_addr + i];
	}
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	/* Implement write operation here */
	gtime++;
	
	if(fs->volume[fp+24] == uchar('r')){
		printf("ERROR: Cannnot implement write on read mode file!\n");
		return -1;
	}
	if(size>1024){
		printf("ERROR: File size exceeds the maximum!\n");
		return -1;
	}
	
	int free_block_offset;
	int free_block_index;
	int block_num = size/32;         //存这个size需要多少的blocks
	int free_bit_map_index;
	if(size%32 != 0){
		block_num += 1;
	}
	if(fs->volume[fp+22]==255){
		//里面还没有存过东西
		//判断还够不够放
		//printf("here!!!\n");
		free_block_offset = find_first_free_block_offset(fs);
		if(fs->volume[fs->SUPERBLOCK_SIZE- (block_num+free_block_offset)*fs->STORAGE_BLOCK_SIZE] != 0){
			printf("ERROR: Space is not enough to write\n");
			return -1;
		}
		//printf("bitsss 1 = %d\n", int(fs->volume[0]));
		//够放置
		//改写bitmap往后面多写，意思是把0替换成1，不用考虑把1换成0的情况
		//printf("block_num 1 = %d\n", block_num);
		change_bitmaps(fs, block_num);
		
		//printf("bitsss 1 = %d\n", int(fs->volume[0]));
		free_block_offset = int((fs->volume[fp+20]<<8)+(fs->volume[fp+21]));
		//printf("free_block_offset is %d\n", free_block_offset);
		free_block_index = fs->FILE_BASE_ADDRESS+ free_block_offset*32;   //第一个空的block的index
		for(int i=0; i<size; i++){
			fs->volume[free_block_index + i] = input[i]; 
		}
		fs->volume[fp+22] = uchar(size>>8);
		fs->volume[fp+23] = uchar(size & 0x000000FF);
		fs->volume[fp + 27] = uchar(gtime >>8);
		fs->volume[fp + 28] = uchar(gtime & 0x000000FF);
		
	}else{
		//里面存过东西了
		//printf("GEEEEEEEEEEEEET IN HERE\n");
		int orig_size = int(((fs->volume[fp+22])<<8)+(fs->volume[fp+23]));
		int block_flag = same_block_size(orig_size,size);
		int block_start_offset = int((fs->volume[fp+20]<<8) + fs->volume[fp+21]);
		int block_start_index = fs->FILE_BASE_ADDRESS + block_start_offset*fs->STORAGE_BLOCK_SIZE;
		//把原来位置上的东西都清掉
		for (int j=0 ; j<orig_size; j++){
			fs->volume[block_start_index + j] = 255;
		}
		//printf("DELETE DONE!\n");
		if(block_flag == 1){
			//printf("YEAH, NO COMPACT!!\n");
			//位置没变，不用compact
			for(int n=0;n<size;n++){
				fs->volume[block_start_index + n] = input[n];
			}			
		}else if(block_flag == 2 || block_flag == 3){
			//printf("ON NO, WE need to compact!\n");
			compact(fs, block_start_offset, block_start_index, orig_size);
			//printf("DID I COME COUT?\n");
			int new_free_block_offset = find_first_free_block_offset(fs);
			
			if(block_flag == 2){
				//orig<curr
				if(size > (fs->STORAGE_SIZE-(new_free_block_offset*32+fs->FILE_BASE_ADDRESS))){
					printf("ERROR: No space for this writern file!\n");
					return -1;
				}	
			}
			fs->volume[fp+20] = uchar(new_free_block_offset>>8);
			fs->volume[fp+21] = uchar(new_free_block_offset & 0x000000FF);
			
			//write
			change_bitmaps(fs, block_num);
			free_block_offset = find_first_free_block_offset(fs);
				free_block_index = fs->FILE_BASE_ADDRESS + free_block_offset*32;
			for(int i=0; i<size; i++){
				fs->volume[free_block_index + i] = input[i]; 
			}
		}
		fs->volume[fp+22] = uchar(size>>8);
		fs->volume[fp+23] = uchar(size & 0x000000FF);
		fs->volume[fp + 27] = uchar(gtime >>8);
		fs->volume[fp + 28] = uchar(gtime & 0x000000FF);
	}
	//printf("\nAfter write function AAAAAAAAAAAAAAAAAA: size is %d\n", int((fs->volume[fp + 22] << 8) + fs->volume[fp + 23]));
	
}
__device__ void fs_gsys(FileSystem *fs, int op)
{
	/* Implement LS_D and LS_S operation here */
	int file_num = bubble_sort(fs,op);
	//printf("filenum = %d\n",file_num);
	int fcb_addr;
	uchar target;
	if(op == LS_D){
		printf("===sort by modified time===\n");
		for (int i=0; i<file_num; i++){
			//printf("here\n");
			fcb_addr = fs->SUPERBLOCK_SIZE + 32*i;
			//printf("fcb_addr is %d\n", fcb_addr);
			for(int j=0; j<20; j++){
				if(int(fs->volume[fcb_addr + j]) == int('\0')){
					printf("\n");
					break;
				}
				printf("%c",uchar(fs->volume[fcb_addr + j]));
			}
		}
	}else if(op == LS_S){
		printf("===sort by file size===\n");
		for (int i=0; i<file_num; i++){
			fcb_addr = fs->SUPERBLOCK_SIZE + 32*i;
			for(int j=0; j<20; j++){
				if(int(fs->volume[fcb_addr + j]) == int('\0')){
					printf(" %d\n",int((fs->volume[fcb_addr+22]<<8) + fs->volume[fcb_addr+23]));
					break;
				}
				printf("%c",uchar(fs->volume[fcb_addr + j]));
			}
		}
	}
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	if(string_len(s)>20){
		printf("ERROR: file name exceeds the maximum!\n");
	}
	/* Implement rm operation here */
	int fcb_index;
	int fcb_find_offset = -3;   //denote the position of the target file in fcb
	int compare_flag = 1;   //1 indicates they are equal, 0 indicates they are different
	if(op == RM){
		//find the file name
		for(int i=0; i<1024;i++){
			compare_flag = 1;
			fcb_index = fs->SUPERBLOCK_SIZE+i*32;
			if(fs->volume[fcb_index] == 255){
				//如果没满 但是一定找不到了
				fcb_find_offset = -2;    //did not find file name s in fcb
				break;
			}else{
				//这个不是空的
				for(int j=0; j< string_len(s);j++){
					if(fs->volume[fcb_index + j] != s[j]){
						compare_flag = 0;
						break;
					}
				}
				if(compare_flag == 1){
					//we find the file name
					fcb_find_offset = i;
					break;
				}
			}
		}
		if(fcb_find_offset==-3 || fcb_find_offset==-2){
			printf("ERROR: Cannnot find the file name in file system!\n");
			return;
		}else if(fcb_find_offset>=0){
			//find file name in fcb
			
			//操作file memory+bitmap
			int fcb_addr = fs->SUPERBLOCK_SIZE + 32*fcb_find_offset;
			int block_start_offset = int((fs->volume[fcb_addr+20]<<8) + fs->volume[fcb_addr+21]);
			int block_start_index = fs->FILE_BASE_ADDRESS + 32*block_start_offset;
			int size = int((fs->volume[fcb_addr+22]<<8) + fs->volume[fcb_addr+23]);
			for(int j=0; j<size; j++){
				fs->volume[block_start_index+j] = 255;
			}
			compact(fs, block_start_offset, block_start_index, size);
			
			
			//操作FCB
			int last_fcb_start_index = find_last_fcb(fs);
			
			for(int y=0; y<32; y++){
				fs->volume[fcb_addr+y] = fs->volume[last_fcb_start_index+y];
			}
			for(int k=0; k<32; k++){
				fs->volume[last_fcb_start_index+k] = 255;
			}
			
		}
		
		
	}
	else{
		printf("ERROR: Invalid instruction!\n");
		return;
	}
}
